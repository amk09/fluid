#include "hip/hip_runtime.h"
#include "fluid_simulation.h"
#include <hip/hip_runtime.h>
#include <>
#include <stdio.h>
#include <vector>
#include <cmath> // For std::floor
#include <algorithm> // For std::swap
#include <numeric>   // For std::accumulate (potential use)

// CUDA error checking macro
#define CUDA_CHECK(call) \
    do { \
        hipError_t error = call; \
        if (error != hipSuccess) { \
            fprintf(stderr, "CUDA error at %s:%d: %s\n", \
                    __FILE__, __LINE__, hipGetErrorString(error)); \
            exit(1); \
        } \
    } while(0)

// Forward declaration for CPU debugging function (using float*)
void CPUlinSolveJacobi(int b, float* x, const float* rhs, float a, float c, int iter, int N);

// Constructor
FluidSimulation::FluidSimulation(int width, int height, int depth) {
    params.width = width;
    params.height = height;
    params.depth = depth;
    params.dt = 0.1f;
    params.visc = 0.0000001f;
    params.diff = 0.00001f;
    
    allocateMemory();
}

// Destructor
FluidSimulation::~FluidSimulation() {
    freeMemory();
}

// Memory allocation
void FluidSimulation::allocateMemory() {
    size_t size = params.width * params.height * params.depth * sizeof(float);
    CUDA_CHECK(hipMalloc(&d_density0, size));
    CUDA_CHECK(hipMalloc(&d_density, size));

    CUDA_CHECK(hipMalloc(&d_Vx, size));
    CUDA_CHECK(hipMalloc(&d_Vy, size));
    CUDA_CHECK(hipMalloc(&d_Vz, size));
    
    CUDA_CHECK(hipMalloc(&d_Vx0, size));
    CUDA_CHECK(hipMalloc(&d_Vy0, size));
    CUDA_CHECK(hipMalloc(&d_Vz0, size));

    CUDA_CHECK(hipMalloc(&d_tmp, size));
    
    resetFields();
}

// Memory cleanup
void FluidSimulation::freeMemory() {
    CUDA_CHECK(hipFree(d_density0));
    CUDA_CHECK(hipFree(d_density));

    CUDA_CHECK(hipFree(d_Vx));
    CUDA_CHECK(hipFree(d_Vy));
    CUDA_CHECK(hipFree(d_Vz));
    
    CUDA_CHECK(hipFree(d_Vx0));
    CUDA_CHECK(hipFree(d_Vy0));
    CUDA_CHECK(hipFree(d_Vz0));

    CUDA_CHECK(hipFree(d_tmp));
}

// Reset all fields to zero
void FluidSimulation::resetFields() {
    size_t size = params.width * params.height * params.depth * sizeof(float);

    CUDA_CHECK(hipMemset(d_density0, 0, size));
    CUDA_CHECK(hipMemset(d_density, 0, size));

    CUDA_CHECK(hipMemset(d_Vx, 0, size));
    CUDA_CHECK(hipMemset(d_Vy, 0, size));
    CUDA_CHECK(hipMemset(d_Vz, 0, size));
    
    CUDA_CHECK(hipMemset(d_Vx0, 0, size));
    CUDA_CHECK(hipMemset(d_Vy0, 0, size));
    CUDA_CHECK(hipMemset(d_Vz0, 0, size));

    CUDA_CHECK(hipMemset(d_tmp, 0, size));

}

// Initialize simulation
void FluidSimulation::initialize() {
    resetFields();
}

// Main simulation step
void FluidSimulation::step() {
    static int current_simulation_step = 0; // Keep track of the simulation step number

    int solver_iterations = 20; // Default iterations for diffuse/project
 
    // Sync threads after each step // This comment seems misplaced, hipDeviceSynchronize is used specifically.
    
    // Velocity Step
    // 1. Diffuse Velocities (input: d_Vx, d_Vy, d_Vz from prev step; output: d_Vx0, d_Vy0, d_Vz0)
    diffuse(1, d_Vx0, d_Vx, params.visc, params.dt, solver_iterations, params.width);
    diffuse(2, d_Vy0, d_Vy, params.visc, params.dt, solver_iterations, params.width);
    diffuse(3, d_Vz0, d_Vz, params.visc, params.dt, solver_iterations, params.width);

    // Project diffused velocities (d_Vx0, d_Vy0, d_Vz0)
    // Using d_Vx as pressure buffer, d_Vy as divergence buffer.
    // Result of projection is in d_Vx0, d_Vy0, d_Vz0.
    project(d_Vx0, d_Vy0, d_Vz0, d_Vx, d_Vy, solver_iterations, params.width);
 
    // 2. Advect Velocities 
    // Input for advection: d_Vx0, d_Vy0, d_Vz0 (diffused and projected)
    // Advecting field: d_Vx0, d_Vy0, d_Vz0
    // Output: d_Vx, d_Vy, d_Vz
    advect(1, d_Vx, d_Vx0, d_Vx0, d_Vy0, d_Vz0, params.dt, params.width);
    advect(2, d_Vy, d_Vy0, d_Vx0, d_Vy0, d_Vz0, params.dt, params.width);
    advect(3, d_Vz, d_Vz0, d_Vx0, d_Vy0, d_Vz0, params.dt, params.width);
 
    // 3. Project advected velocities (d_Vx, d_Vy, d_Vz)
    // Using d_Vx0 as pressure buffer, d_Vy0 as divergence buffer.
    // Result of projection is in d_Vx, d_Vy, d_Vz.
    project(d_Vx, d_Vy, d_Vz, d_Vx0, d_Vy0, solver_iterations, params.width);

    // Density Step
    // 1. Diffuse Density (input: d_density from prev step; output: d_density0)
    //printf("[Step %d] Density sum before diffuse: %f (reading from d_density)\n", current_simulation_step, sumFieldHost(d_density));
    diffuse(0, d_density0, d_density, params.diff, params.dt, solver_iterations, params.width);
    //printf("[Step %d] Density sum after diffuse:  %f (result in d_density0)\n", current_simulation_step, sumFieldHost(d_density0));
    
    // 2. Advect Density 
    // Input for advection: d_density0 (diffused density)
    // Advecting field: d_Vx, d_Vy, d_Vz (divergence-free)
    // Output: d_density
    // printf("[Step %d] Density sum before advect: %f (reading from d_density0)\n", current_simulation_step, sumFieldHost(d_density0)); // Optional, should be same as after diffuse
    advect(0, d_density, d_density0, d_Vx, d_Vy, d_Vz, params.dt, params.width);
    CUDA_CHECK(hipDeviceSynchronize());
    //printf("[Step %d] Density sum after advect:   %f (result in d_density)\n", current_simulation_step, sumFieldHost(d_density));

    current_simulation_step++;
}

// Device version of IX function for CUDA kernels
__device__ int IX_device(int x, int y, int z, int width, int height) {
    return x + width * (y + height * z);  // x-major order
}

// Host version of IX function
int FluidSimulation::IX(int x, int y, int z) {
    return x + params.width * (y + params.height * z);  // x-major order
}

// CUDA kernel for adding velocity
__global__ void addVelocityKernel(float* d_Vx, float* d_Vy, float* d_Vz, 
                                int x, int y, int z, float vx, float vy, float vz,
                                int width, int height, int depth) {
    int idx = (z * height + y) * width + x;
    if (x >= 0 && x < width && y >= 0 && y < height && z >= 0 && z < depth) {
        d_Vx[idx] += vx;
        d_Vy[idx] += vy;
        d_Vz[idx] += vz;
    }
}

// CUDA kernel for adding density
__global__ void addDensityKernel(float* d_density, int x, int y, int z, 
                               float amount, int width, int height, int depth) {
    int idx = (z * height + y) * width + x;
    if (x >= 0 && x < width && y >= 0 && y < height && z >= 0 && z < depth) {
        d_density[idx] += amount;
    }
}

__device__ float computeGaussianWeightForOffset(int dx, int dy, int dz, float sigma) {
    if (sigma < 1e-6f) { // Treat very small sigma as effectively zero
        return (dx == 0 && dy == 0 && dz == 0) ? 1.0f : 0.0f;
    }
    float variance = sigma * sigma;
    // dx, dy, dz are integer offsets. Convert to float for calculation.
    float fdx = static_cast<float>(dx);
    float fdy = static_cast<float>(dy);
    float fdz = static_cast<float>(dz);
    float r_squared = fdx*fdx + fdy*fdy + fdz*fdz;
    // Ensure variance is not zero to prevent division by zero if sigma was extremely small but not < 1e-6f
    if (variance < 1e-9f) { // A very small positive variance
         return (r_squared < 1e-9f) ? 1.0f : 0.0f; // Effectively 1 if at center, 0 otherwise
    }
    return expf(-r_squared / (2.0f * variance));
}

__global__ void addDensityGaussianKernel(float* d_density,
                                       float centerX, float centerY, float centerZ,
                                       float amount, float sigma, int radius,
                                       int width, int height, int depth) {
    // Each thread handles one combination of (dx_offset, dy_offset, dz_offset)
    // where offsets range from -radius to +radius.
    // Total span for iteration is 2*radius + 1.
    int span = 2 * radius + 1;

    int flat_idx_x = blockIdx.x * blockDim.x + threadIdx.x;
    int flat_idx_y = blockIdx.y * blockDim.y + threadIdx.y;
    int flat_idx_z = blockIdx.z * blockDim.z + threadIdx.z;

    if (flat_idx_x >= span || flat_idx_y >= span || flat_idx_z >= span) {
        return; // This thread is outside the required (0 to span-1) range
    }

    int dx_offset = flat_idx_x - radius; // Converts flat_idx (0 to span-1) to offset (-radius to +radius)
    int dy_offset = flat_idx_y - radius;
    int dz_offset = flat_idx_z - radius;

    // Calculate target cell coordinates based on float center and integer offset, truncating like static_cast<int>
    int nx = static_cast<int>(centerX + static_cast<float>(dx_offset));
    int ny = static_cast<int>(centerY + static_cast<float>(dy_offset));
    int nz = static_cast<int>(centerZ + static_cast<float>(dz_offset));

    // Check if the target cell is within the simulation grid
    if (nx < 0 || nx >= width || ny < 0 || ny >= height || nz < 0 || nz >= depth) {
        return;
    }

    // Calculate Gaussian weight using the integer offsets
    float weight = computeGaussianWeightForOffset(dx_offset, dy_offset, dz_offset, sigma);

    if (weight > 1e-6f) { // Add only if there's a non-negligible contribution
        float val_to_add = amount * weight * 0.7f; // As per user's example
        atomicAdd(&d_density[IX_device(nx, ny, nz, width, height)], val_to_add);
    }
}

// Get density field as host vector
std::vector<float> FluidSimulation::getDensityFieldHost() {
    size_t size = params.width * params.height * params.depth;
    std::vector<float> host_data(size);
    CUDA_CHECK(hipMemcpy(host_data.data(), d_density, size * sizeof(float), hipMemcpyDeviceToHost));
    return host_data;
}

// Get velocity field as host vector
std::vector<float> FluidSimulation::getVelocityFieldHost(int component) {
    size_t size = params.width * params.height * params.depth;
    std::vector<float> host_data(size);
    
    float* d_field;
    switch(component) {
        case 0: d_field = d_Vx; break;
        case 1: d_field = d_Vy; break;
        case 2: d_field = d_Vz; break;
        default: return std::vector<float>(); // Return empty vector for invalid component
    }
    
    CUDA_CHECK(hipMemcpy(host_data.data(), d_field, size * sizeof(float), hipMemcpyDeviceToHost));
    return host_data;
}

// Add velocity at a point (using CUDA kernel)
void FluidSimulation::addVelocity(int x, int y, int z, float vx, float vy, float vz) {
    if (x < 0 || x >= params.width || y < 0 || y >= params.height || z < 0 || z >= params.depth) return;
    
    dim3 block(1);
    dim3 grid(1);
    addVelocityKernel<<<grid, block>>>(d_Vx, d_Vy, d_Vz, x, y, z, vx, vy, vz,
                                      params.width, params.height, params.depth);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());
}

// Add density at a point (using CUDA kernel)
void FluidSimulation::addDensity(int x, int y, int z, float amount) {
    if (x < 0 || x >= params.width || y < 0 || y >= params.height || z < 0 || z >= params.depth) return;
    
    dim3 block(1);
    dim3 grid(1);
    addDensityKernel<<<grid, block>>>(d_density, x, y, z, amount,
                                     params.width, params.height, params.depth);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());
    
    // Debug print
    printf("Added density %f at (%d,%d,%d)\n", amount, x, y, z);
}

// Add density with a Gaussian distribution
void FluidSimulation::addDensityGaussian(float centerX, float centerY, float centerZ, float amount, float sigma) {
    if (sigma < 0.0f) { // Sigma should be non-negative
        sigma = 0.0f;
    }

    // Determine the radius of influence for the Gaussian, as per user's example snippet
    // The loop iterates from -radius to +radius for offsets.
    int radius = static_cast<int>(sigma * 2.0f);
    if (radius < 0) radius = 0; 

    // The kernel iterates for offsets from -radius to +radius.
    // The total number of iterations (span) in each dimension is 2*radius + 1.
    int span = 2 * radius + 1;
    // If sigma is 0, radius is 0, span is 1. This is correct for a single point.

    dim3 threadsPerBlock(8, 8, 4); // Example block size
    dim3 numBlocks(
        (span + threadsPerBlock.x - 1) / threadsPerBlock.x,
        (span + threadsPerBlock.y - 1) / threadsPerBlock.y,
        (span + threadsPerBlock.z - 1) / threadsPerBlock.z
    );
    
    // Ensure numBlocks components are at least 1 if span is 1.
    if (numBlocks.x == 0) numBlocks.x = 1;
    if (numBlocks.y == 0) numBlocks.y = 1;
    if (numBlocks.z == 0) numBlocks.z = 1;


    addDensityGaussianKernel<<<numBlocks, threadsPerBlock>>>(
        d_density, // Target the main density buffer
        centerX, centerY, centerZ, amount, sigma, radius,
        params.width, params.height, params.depth
    );
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize()); // Ensure kernel completes and effect is visible
}

// Get current density field
float* FluidSimulation::getDensityField() {
    return d_density;
}

// Save grid data to files
void FluidSimulation::saveGridData(int step) {
    // Create plots directory if it doesn't exist
    system("mkdir -p ../visualizer/renderData");
    
    // Get dimensions
    int N = params.width;
    size_t size = N * N * N * sizeof(float);
    
    // Create temporary arrays for CPU
    std::vector<float> density(N * N * N);
    std::vector<float> velocityX(N * N * N);
    std::vector<float> velocityY(N * N * N);
    std::vector<float> velocityZ(N * N * N);
    std::vector<float> velocityMag(N * N * N);
    
    // Copy data from GPU to CPU
    CUDA_CHECK(hipMemcpy(density.data(), d_density, size, hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(velocityX.data(), d_Vx, size, hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(velocityY.data(), d_Vy, size, hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(velocityZ.data(), d_Vz, size, hipMemcpyDeviceToHost));
    // Create filename with step number
    char filename[256];
    
    // Save density
    snprintf(filename, sizeof(filename), "../visualizer/renderData/density_%04d.bin", step);
    FILE* fp = fopen(filename, "wb");
    if (fp) {
        // Write dimensions first
        // fwrite(&N, sizeof(int), 1, fp);
        // fwrite(&N, sizeof(int), 1, fp);
        // fwrite(&N, sizeof(int), 1, fp);
        // Write density data
        fwrite(density.data(), sizeof(float), N * N * N, fp);
        fclose(fp);
    }

    // Save density
    snprintf(filename, sizeof(filename), "../visualizer/renderData/color_%04d.bin", step);
    fp = fopen(filename, "wb");
    if (fp) {
        // Write dimensions first
        // fwrite(&N, sizeof(int), 1, fp);
        // fwrite(&N, sizeof(int), 1, fp);
        // fwrite(&N, sizeof(int), 1, fp);
        // Write density data
        fwrite(density.data(), sizeof(float), N * N * N, fp);
        fclose(fp);
    }
    
    // // Save velocity X
    // snprintf(filename, sizeof(filename), "../visualizer/renderData/velocityX_%04d.bin", step);
    // fp = fopen(filename, "wb");
    // if (fp) {
    //     // Write dimensions first
    //     fwrite(&N, sizeof(int), 1, fp);
    //     fwrite(&N, sizeof(int), 1, fp);
    //     fwrite(&N, sizeof(int), 1, fp);
    //     // Write velocity data
    //     fwrite(velocityX.data(), sizeof(float), N * N * N, fp);
    //     fclose(fp);
    // }
    
    // // Save velocity Y
    // snprintf(filename, sizeof(filename), "../visualizer/renderData/velocityY_%04d.bin", step);
    // fp = fopen(filename, "wb");
    // if (fp) {
    //     // Write dimensions first
    //     fwrite(&N, sizeof(int), 1, fp);
    //     fwrite(&N, sizeof(int), 1, fp);
    //     fwrite(&N, sizeof(int), 1, fp);
    //     // Write velocity data
    //     fwrite(velocityY.data(), sizeof(float), N * N * N, fp);
    //     fclose(fp);
    // }
    
    // // Save velocity Z
    // snprintf(filename, sizeof(filename), "../visualizer/renderData/velocityZ_%04d.bin", step);
    // fp = fopen(filename, "wb");
    // if (fp) {
    //     // Write dimensions first
    //     fwrite(&N, sizeof(int), 1, fp);
    //     fwrite(&N, sizeof(int), 1, fp);
    //     fwrite(&N, sizeof(int), 1, fp);
    //     // Write velocity data
    //     fwrite(velocityZ.data(), sizeof(float), N * N * N, fp);
    //     fclose(fp);
    // }
    // Save velocity magnitude
    snprintf(filename, sizeof(filename), "../visualizer/renderData/velocity_%04d.bin", step);
    fp = fopen(filename, "wb");
    if (fp) {
        // Write dimensions first
        // fwrite(&N, sizeof(int), 1, fp); 
        // fwrite(&N, sizeof(int), 1, fp);
        // fwrite(&N, sizeof(int), 1, fp);

        // Calculate velocity magnitude
        for (int i = 0; i < N * N * N; i++) {
            velocityMag[i] = sqrt(velocityX[i] * velocityX[i] + velocityY[i] * velocityY[i] + velocityZ[i] * velocityZ[i]);
        }

        // Write velocity data
        fwrite(velocityMag.data(), sizeof(float), N * N * N, fp);
        fclose(fp);
    }   
    

    printf("Saved grid data to file \"../visualizer/renderData/dataZ_%04d.bin\"\n", step);
}

// Simulation Methods:

// Base Methods:

// CUDA kernel for linear solver
// __global__ void linSolveKernel(float* x, float* x0, float a, float c, int N) {
//     int i = blockIdx.x * blockDim.x + threadIdx.x + 1;
//     int j = blockIdx.y * blockDim.y + threadIdx.y + 1;
//     int k = blockIdx.z * blockDim.z + threadIdx.z + 1;
    
//     if (i < N-1 && j < N-1 && k < N-1) {
//         float cRecip = 1.0f / c;
        
//         // Get current cell value
//         float current = x0[IX_device(i, j, k, N, N)];
        
//         // Get neighboring cell values
//         float x_plus = x[IX_device(i+1, j, k, N, N)];
//         float x_minus = x[IX_device(i-1, j, k, N, N)];
//         float y_plus = x[IX_device(i, j+1, k, N, N)];
//         float y_minus = x[IX_device(i, j-1, k, N, N)];
//         float z_plus = x[IX_device(i, j, k+1, N, N)];
//         float z_minus = x[IX_device(i, j, k-1, N, N)];
        
//         // Calculate new value
//         float new_val = (current + a * (x_plus + x_minus + y_plus + y_minus + z_plus + z_minus)) * cRecip;
        
//         // Store result
//         x[IX_device(i, j, k, N, N)] = new_val;
//     }
// }

// // Linear solver for diffusion and pressure
// void FluidSimulation::linSolve(int b, float* x, float* x0, float a, float c, int iter, int N) {
//     // Calculate grid and block dimensions
//     dim3 blockDim(8, 8, 8);  // 8x8x8 threads per block
//     dim3 gridDim(
//         (N + blockDim.x - 1) / blockDim.x,
//         (N + blockDim.y - 1) / blockDim.y,
//         (N + blockDim.z - 1) / blockDim.z
//     );
    
//     // Perform iterations
//     for (int k = 0; k < iter; k++) {
//         // Launch kernel
//         linSolveKernel<<<gridDim, blockDim>>>(x, x0, a, c, N);
//         CUDA_CHECK(hipGetLastError());
//         CUDA_CHECK(hipDeviceSynchronize());
        
//         // Set boundary conditions
//         set_bnd(b, x, N);
//     }
// }

#define RADIUS 1                     // Stencil radius (1 for 7-point Laplace)
#define BLOCK_X 8                    // Threads per block in X
#define BLOCK_Y 8                    // Threads per block in Y
#define BLOCK_Z 8                    // Threads per block in Z
//
// Shared-memory footprint = (BLOCK_Z+2R) * (BLOCK_Y+2R) * (BLOCK_X+2R) floats
// → (8+2)³ = 1000 floats ≈ 4 kB  → plenty of occupancy headroom on most GPUs
// -----------------------------------------------------------------------------

__global__ void jacobiSweep3D_shared(float*       __restrict__ dst,
                                     const float* __restrict__ src,
                                     const float* __restrict__ rhs,
                                     float                     a,
                                     float                     cRecip,
                                     int                       N)
{
    // ----------------------------- Global coordinates ------------------------
    int gi = blockIdx.x * BLOCK_X + threadIdx.x;           // 0 … N-1
    int gj = blockIdx.y * BLOCK_Y + threadIdx.y;
    int gk = blockIdx.z * BLOCK_Z + threadIdx.z;

    // Skip the outer one-cell frame; guards later assume 1 ≤ gi<N-1, etc.
    if (gi >= N || gj >= N || gk >= N) return;

    // ----------------------------- Shared memory tile ------------------------
    extern __shared__ float sh[];   // 3-D slab, flattened
    // Dimensions inside shared memory
    const int shX = BLOCK_X + 2*RADIUS;
    const int shY = BLOCK_Y + 2*RADIUS;
    const int shZ = BLOCK_Z + 2*RADIUS;

    // Lambda for flattening (z,y,x) → 1-D
    auto sidx = [=] __device__ (int z,int y,int x) {
        return (z*shY + y)*shX + x;
    };

    // Local coords *inside* the shared tile (including halo offset)
    int li = threadIdx.x + RADIUS;
    int lj = threadIdx.y + RADIUS;
    int lk = threadIdx.z + RADIUS;

    // ----------------------------- Load centre cell --------------------------
    sh[sidx(lk, lj, li)] = src[IX_device(gi, gj, gk, N, N)];

    // ----------------------------- Load halo cells ---------------------------
    // Every thread cooperatively pulls in at most 6 neighbours (faces only).
    // We keep it branch-free; if a neighbour lies outside the true domain we
    // re-use the centre value (Dirichlet zero-gradient boundary).

    // Offsets for 6 face neighbours
#pragma unroll
    for (int face = 0; face < 6; ++face) {
        int di = (face == 0) - (face == 1);   // +1 x, −1 x
        int dj = (face == 2) - (face == 3);   // +1 y, −1 y
        int dk = (face == 4) - (face == 5);   // +1 z, −1 z

        int gni = gi + di;
        int gnj = gj + dj;
        int gnk = gk + dk;

        // Position in shared memory for that neighbour
        int lni = li + di;
        int lnj = lj + dj;
        int lnk = lk + dk;

        // Bounds check once (all threads do identical comparisons → no warp diverge)
        bool inside = (gni >= 0 && gni < N &&
                       gnj >= 0 && gnj < N &&
                       gnk >= 0 && gnk < N);

        sh[sidx(lnk, lnj, lni)] =
            inside ? src[IX_device(gni, gnj, gnk, N, N)]
                   : sh[sidx(lk, lj, li)];  // replicate centre for out-of-domain
    }
    __syncthreads();

    // ----------------------------- Compute Jacobi update ---------------------
    // Skip the global outer frame to avoid reading unallocated memory
    if (gi > 0 && gj > 0 && gk > 0 && gi < N-1 && gj < N-1 && gk < N-1) {

        float nbrSum =
              sh[sidx(lk,     lj,     li+1)] + sh[sidx(lk,     lj,     li-1)]
            + sh[sidx(lk,     lj+1,   li  )] + sh[sidx(lk,     lj-1,   li  )]
            + sh[sidx(lk+1,   lj,     li  )] + sh[sidx(lk-1,   lj,     li  )];

        int gIdx = IX_device(gi, gj, gk, N, N);
        dst[gIdx] = (rhs[gIdx] + a * nbrSum) * cRecip;
    }
}

__global__ void jacobiSweep3D(float*       dst,
                              const float* src,
                              const float* rhs,
                              float        a,
                              float        cRecip,
                              int          N)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x + 1;
    int j = blockIdx.y * blockDim.y + threadIdx.y + 1;
    int k = blockIdx.z * blockDim.z + threadIdx.z + 1;
    if (i >= N-1 || j >= N-1 || k >= N-1) return;

    int idx = IX_device(i,j,k,N,N);

    // Sum neighbors from the source array (previous iteration)
    float nbrSum =  src[IX_device(i+1,j  ,k  ,N,N)] + src[IX_device(i-1,j  ,k  ,N,N)]
                  + src[IX_device(i  ,j+1,k  ,N,N)] + src[IX_device(i  ,j-1,k  ,N,N)]
                  + src[IX_device(i  ,j  ,k+1,N,N)] + src[IX_device(i  ,j  ,k-1,N,N)];

    // Calculate new value and write to destination array
    dst[idx] = (rhs[idx] + a * nbrSum) * cRecip;
}

// // host wrapper for Jacobi solver
void FluidSimulation::linSolve(int    b,        // field type 0,1,2,3 for boundaries
                                       float* x,        // IN: initial guess, OUT: solution
                                       float* rhs,      // immutable rhs (like divergence)
                                       float  a, float c,
                                       int    iter, int N)
{
    // Use the pre-allocated temporary buffer
    float* tmp = d_tmp;          // same size as x

    float* src = x;              // start reading from x
    float* dst = tmp;            // write into tmp first

    dim3 blk(8,8,8);
    // Grid dimensions cover the interior cells (1 to N-2)
    dim3 grd( (N-2 + blk.x-1) / blk.x,
              (N-2 + blk.y-1) / blk.y,
              (N-2 + blk.z-1) / blk.z );

    float cRecip = 1.0f / c;

    for (int n = 0; n < iter; ++n) {
        // Perform one Jacobi sweep
        // dim3 block(BLOCK_X, BLOCK_Y, BLOCK_Z);

        // // Domain interior is (N-2)³ points (we skip the boundaries),
        // // but we can simply launch enough blocks to cover the full N³
        // // cube and rely on the i>0 && i<N-1 guard inside the kernel.
        // dim3 grid( (N + BLOCK_X - 1) / BLOCK_X,
        //         (N + BLOCK_Y - 1) / BLOCK_Y,
        //         (N + BLOCK_Z - 1) / BLOCK_Z );

        // size_t shMemBytes = (BLOCK_X + 2*RADIUS) *
        //                     (BLOCK_Y + 2*RADIUS) *
        //                     (BLOCK_Z + 2*RADIUS) * sizeof(float);

        // jacobiSweep3D_shared<<<grid, block, shMemBytes>>>(
        //         dst, src, rhs, a, cRecip, N);
        jacobiSweep3D<<<grd, blk>>>(dst, src, rhs, a, cRecip, N);
        CUDA_CHECK(hipGetLastError());
        // No synchronize needed here, boundary kernel launch will sync.

        // Enforce boundary conditions on the *newly written* data in 'dst'
        set_bnd(b, dst, N);

        // Swap pointers for next iteration:
        // the destination of this step becomes the source for the next
        std::swap(src, dst);
    }

    // After the loop, 'src' points to the array holding the final result
    // (because of the last swap). If 'src' is not the original 'x' array,
    // we need to copy the result back into 'x'.
    if (src != x) {
        size_t total_elements = (size_t)N * N * N;
        size_t size_bytes = total_elements * sizeof(float);
        CUDA_CHECK(hipMemcpy(x, src, size_bytes, hipMemcpyDeviceToDevice));
    }
}

// CUDA kernel for setting face boundaries
__global__ void set_bnd_kernel(float* x, int b, int N) {
    // Handle X-Face boundaries (i=0 and i=N-1)
    if (blockIdx.z == 0) {
        int j = blockIdx.x * blockDim.x + threadIdx.x + 1;
        int k = blockIdx.y * blockDim.y + threadIdx.y + 1;
        
        if (j < N-1 && k < N-1) {
            if (b == 1) {
                x[IX_device(0, j, k, N, N)] = -x[IX_device(1, j, k, N, N)];
                x[IX_device(N-1, j, k, N, N)] = -x[IX_device(N-2, j, k, N, N)];
            } else {
                x[IX_device(0, j, k, N, N)] = x[IX_device(1, j, k, N, N)];
                x[IX_device(N-1, j, k, N, N)] = x[IX_device(N-2, j, k, N, N)];
            }
        }
    }
    
    // Handle Y-Face boundaries (j=0 and j=N-1)
    if (blockIdx.z == 1) {
        int i = blockIdx.x * blockDim.x + threadIdx.x + 1;
        int k = blockIdx.y * blockDim.y + threadIdx.y + 1;
        
        if (i < N-1 && k < N-1) {
            if (b == 2) {
                x[IX_device(i, 0, k, N, N)] = -x[IX_device(i, 1, k, N, N)];
                x[IX_device(i, N-1, k, N, N)] = -x[IX_device(i, N-2, k, N, N)];
            } else {
                x[IX_device(i, 0, k, N, N)] = x[IX_device(i, 1, k, N, N)];
                x[IX_device(i, N-1, k, N, N)] = x[IX_device(i, N-2, k, N, N)];
            }
        }
    }
    
    // Handle Z-Face boundaries (k=0 and k=N-1)
    if (blockIdx.z == 2) {
        int i = blockIdx.x * blockDim.x + threadIdx.x + 1;
        int j = blockIdx.y * blockDim.y + threadIdx.y + 1;
        
        if (i < N-1 && j < N-1) {
            if (b == 3) {
                x[IX_device(i, j, 0, N, N)] = -x[IX_device(i, j, 1, N, N)];
                x[IX_device(i, j, N-1, N, N)] = -x[IX_device(i, j, N-2, N, N)];
            } else {
                x[IX_device(i, j, 0, N, N)] = x[IX_device(i, j, 1, N, N)];
                x[IX_device(i, j, N-1, N, N)] = x[IX_device(i, j, N-2, N, N)];
            }
        }
    }
}

// CUDA kernel for setting corner boundaries
__global__ void set_bnd_corner_kernel(float* x, int N) {
    // Only one thread needed for corners
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        // Corner (0,0,0)
        x[IX_device(0, 0, 0, N, N)] = 0.33f * (x[IX_device(1, 0, 0, N, N)] + x[IX_device(0, 1, 0, N, N)] + x[IX_device(0, 0, 1, N, N)]);
        
        // Corner (0,N-1,0)
        x[IX_device(0, N-1, 0, N, N)] = 0.33f * (x[IX_device(1, N-1, 0, N, N)] + x[IX_device(0, N-2, 0, N, N)] + x[IX_device(0, N-1, 1, N, N)]);
        
        // Corner (0,0,N-1)
        x[IX_device(0, 0, N-1, N, N)] = 0.33f * (x[IX_device(1, 0, N-1, N, N)] + x[IX_device(0, 1, N-1, N, N)] + x[IX_device(0, 0, N-2, N, N)]);
        
        // Corner (0,N-1,N-1)
        x[IX_device(0, N-1, N-1, N, N)] = 0.33f * (x[IX_device(1, N-1, N-1, N, N)] + x[IX_device(0, N-2, N-1, N, N)] + x[IX_device(0, N-1, N-2, N, N)]);
        
        // Corner (N-1,0,0)
        x[IX_device(N-1, 0, 0, N, N)] = 0.33f * (x[IX_device(N-2, 0, 0, N, N)] + x[IX_device(N-1, 1, 0, N, N)] + x[IX_device(N-1, 0, 1, N, N)]);
        
        // Corner (N-1,N-1,0)
        x[IX_device(N-1, N-1, 0, N, N)] = 0.33f * (x[IX_device(N-2, N-1, 0, N, N)] + x[IX_device(N-1, N-2, 0, N, N)] + x[IX_device(N-1, N-1, 1, N, N)]);
        
        // Corner (N-1,0,N-1)
        x[IX_device(N-1, 0, N-1, N, N)] = 0.33f * (x[IX_device(N-2, 0, N-1, N, N)] + x[IX_device(N-1, 1, N-1, N, N)] + x[IX_device(N-1, 0, N-2, N, N)]);
        
        // Corner (N-1,N-1,N-1)
        x[IX_device(N-1, N-1, N-1, N, N)] = 0.33f * (x[IX_device(N-2, N-1, N-1, N, N)] + x[IX_device(N-1, N-2, N-1, N, N)] + x[IX_device(N-1, N-1, N-2, N, N)]);
    }
}

// Set boundary conditions
void FluidSimulation::set_bnd(int b, float* x, int N) {
    // Calculate grid and block dimensions for face boundaries
    dim3 blockDim(16, 16);
    dim3 gridDim(
        (N + blockDim.x - 1) / blockDim.x,
        (N + blockDim.y - 1) / blockDim.y,
        3  // 3 types of faces: X, Y, Z
    );
    
    // Set face boundaries
    set_bnd_kernel<<<gridDim, blockDim>>>(x, b, N);
    CUDA_CHECK(hipGetLastError());

    // Synchronize to ensure all operations are complete
    CUDA_CHECK(hipDeviceSynchronize());

    // Set corner boundaries
    set_bnd_corner_kernel<<<1, 1>>>(x, N);
    CUDA_CHECK(hipGetLastError());
    
    // Synchronize to ensure all operations are complete
    CUDA_CHECK(hipDeviceSynchronize());
}

// Set corner boundaries
void FluidSimulation::set_bnd_corner(float* x, int N) {
    set_bnd_corner_kernel<<<1, 1>>>(x, N);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());
}

// Diffusion step
void FluidSimulation::diffuse(int b, float* x, float* x0, float diff, float dt, int iter, int N) {
    float a = dt * diff * (N - 2) * (N - 2);
    linSolve(b, x, x0, a, 1 + 6 * a, iter, N);
}

// CUDA kernel for advection
__global__ void advectKernel(float* d, float* d0, float* velocX, float* velocY, float* velocZ, 
                            float dt, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x + 1;
    int j = blockIdx.y * blockDim.y + threadIdx.y + 1;
    int k = blockIdx.z * blockDim.z + threadIdx.z + 1;
    
    if (i < N-1 && j < N-1 && k < N-1) {
        float dtx = dt * (N - 2);
        float dty = dt * (N - 2);
        float dtz = dt * (N - 2);
        
        // Get velocity at current position
        float tmp1 = dtx * velocX[IX_device(i, j, k, N, N)];
        float tmp2 = dty * velocY[IX_device(i, j, k, N, N)];
        float tmp3 = dtz * velocZ[IX_device(i, j, k, N, N)];
        
        // Calculate back-traced position
        float x = (float)i - tmp1;
        float y = (float)j - tmp2;
        float z = (float)k - tmp3;
        
        // Clamp positions to valid range for interpolation
        float Nfloat = (float)N;
        if(x < 0.5f) x = 0.5f; 
        if(x > Nfloat - 1.5f) x = Nfloat - 1.5f; 
        if(y < 0.5f) y = 0.5f; 
        if(y > Nfloat - 1.5f) y = Nfloat - 1.5f; 
        if(z < 0.5f) z = 0.5f;
        if(z > Nfloat - 1.5f) z = Nfloat - 1.5f;
        
        // Get integer positions for interpolation
        int i0 = floorf(x);
        int i1 = i0 + 1;
        int j0 = floorf(y);
        int j1 = j0 + 1;
        int k0 = floorf(z);
        int k1 = k0 + 1;
        
        // Calculate interpolation weights
        float s1 = x - i0;
        float s0 = 1.0f - s1;
        float t1 = y - j0;
        float t0 = 1.0f - t1;
        float u1 = z - k0;
        float u0 = 1.0f - u1;
        
        // Perform trilinear interpolation
        d[IX_device(i, j, k, N, N)] = 
            s0 * (t0 * (u0 * d0[IX_device(i0, j0, k0, N, N)] +
                        u1 * d0[IX_device(i0, j0, k1, N, N)]) +
                  t1 * (u0 * d0[IX_device(i0, j1, k0, N, N)] +
                        u1 * d0[IX_device(i0, j1, k1, N, N)])) +
            s1 * (t0 * (u0 * d0[IX_device(i1, j0, k0, N, N)] +
                        u1 * d0[IX_device(i1, j0, k1, N, N)]) +
                  t1 * (u0 * d0[IX_device(i1, j1, k0, N, N)] +
                        u1 * d0[IX_device(i1, j1, k1, N, N)]));
    }
}

// Advection step
void FluidSimulation::advect(int b, float* d, float* d0, float* velocX, float* velocY, float* velocZ, float dt, int N) {
    // Calculate grid and block dimensions
    dim3 blockDim(8, 8, 8);  // 8x8x8 threads per block
    dim3 gridDim(
        (N + blockDim.x - 1) / blockDim.x,
        (N + blockDim.y - 1) / blockDim.y,
        (N + blockDim.z - 1) / blockDim.z
    );
    
    // Launch kernel
    advectKernel<<<gridDim, blockDim>>>(d, d0, velocX, velocY, velocZ, dt, N);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());
    
    // Set boundary conditions
    set_bnd(b, d, N);
}

// Helper function to sum field values on the host (for debugging)
float FluidSimulation::sumFieldHost(float* d_field) {
    size_t num_elements = params.width * params.height * params.depth;
    size_t size_bytes = num_elements * sizeof(float);
    std::vector<float> h_field(num_elements);

    CUDA_CHECK(hipMemcpy(h_field.data(), d_field, size_bytes, hipMemcpyDeviceToHost));

    float sum = 0.0f;
    for (size_t i = 0; i < num_elements; ++i) {
        sum += h_field[i];
    }
    return sum;
}

__global__ void computeDivergenceKernel(float* d_divergence, float* d_pressure,
                                      float* d_Vx, float* d_Vy, float* d_Vz, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x + 1;
    int j = blockIdx.y * blockDim.y + threadIdx.y + 1;
    int k = blockIdx.z * blockDim.z + threadIdx.z + 1;

    if (i < N - 1 && j < N - 1 && k < N - 1) {
        int idx = IX_device(i, j, k, N, N);
        
        float div_val = -0.5f * (
            (d_Vx[IX_device(i + 1, j, k, N, N)] - d_Vx[IX_device(i - 1, j, k, N, N)]) +
            (d_Vy[IX_device(i, j + 1, k, N, N)] - d_Vy[IX_device(i, j - 1, k, N, N)]) +
            (d_Vz[IX_device(i, j, k + 1, N, N)] - d_Vz[IX_device(i, j, k - 1, N, N)])
        ) / N; // As per user's provided formula

        d_divergence[idx] = div_val;
        d_pressure[idx] = 0.0f; // Initialize pressure to zero
    }
}

__global__ void subtractPressureGradientKernel(float* d_Vx, float* d_Vy, float* d_Vz,
                                             float* d_pressure, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x + 1;
    int j = blockIdx.y * blockDim.y + threadIdx.y + 1;
    int k = blockIdx.z * blockDim.z + threadIdx.z + 1;

    if (i < N - 1 && j < N - 1 && k < N - 1) {
        int idx = IX_device(i, j, k, N, N);

        d_Vx[idx] -= 0.5f * (d_pressure[IX_device(i + 1, j, k, N, N)] - d_pressure[IX_device(i - 1, j, k, N, N)]) * N;
        d_Vy[idx] -= 0.5f * (d_pressure[IX_device(i, j + 1, k, N, N)] - d_pressure[IX_device(i, j - 1, k, N, N)]) * N;
        d_Vz[idx] -= 0.5f * (d_pressure[IX_device(i, j, k + 1, N, N)] - d_pressure[IX_device(i, j, k - 1, N, N)]) * N;
    }
}

// Projection step: Enforces divergence-free condition on velocity field
void FluidSimulation::project(float *velocX, float *velocY, float *velocZ, float *p_buffer, float *div_buffer, int iter, int N) {
    // Calculate grid and block dimensions
    dim3 blockDim(8, 8, 8);
    dim3 gridDim(
        (N + blockDim.x - 1) / blockDim.x,
        (N + blockDim.y - 1) / blockDim.y,
        (N + blockDim.z - 1) / blockDim.z
    );

    // Step 1: Compute divergence and initialize pressure buffer to 0
    computeDivergenceKernel<<<gridDim, blockDim>>>(div_buffer, p_buffer, velocX, velocY, velocZ, N);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    set_bnd(0, div_buffer, N); // Boundary condition for divergence
    set_bnd(0, p_buffer, N);   // Boundary condition for pressure (initial guess)
    CUDA_CHECK(hipDeviceSynchronize()); // Ensure BCs are set before copying
    // Note: linSolve modifies the 'x' buffer in place (p_buffer here)
    linSolve(0, p_buffer, div_buffer, 1.0f, 6.0f, iter, N);

    // --- Continue with original project logic ---

    // Step 3: Subtract the pressure gradient from the velocity field
    // Note: This uses the p_buffer which now contains the GPU result
    subtractPressureGradientKernel<<<gridDim, blockDim>>>(velocX, velocY, velocZ, p_buffer, N);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    set_bnd(1, velocX, N); // Boundary conditions for velocity components
    set_bnd(2, velocY, N);
    set_bnd(3, velocZ, N);
}

__global__ void addVelocityGaussianKernel(float* d_Vx, float* d_Vy, float* d_Vz,
                                       float centerX, float centerY, float centerZ,
                                       float amountVx, float amountVy, float amountVz, float sigma, int radius,
                                       int width, int height, int depth) {
    // Each thread handles one combination of (dx_offset, dy_offset, dz_offset)
    int span = 2 * radius + 1;

    int flat_idx_x = blockIdx.x * blockDim.x + threadIdx.x;
    int flat_idx_y = blockIdx.y * blockDim.y + threadIdx.y;
    int flat_idx_z = blockIdx.z * blockDim.z + threadIdx.z;

    if (flat_idx_x >= span || flat_idx_y >= span || flat_idx_z >= span) {
        return;
    }

    int dx_offset = flat_idx_x - radius;
    int dy_offset = flat_idx_y - radius;
    int dz_offset = flat_idx_z - radius;

    int nx = static_cast<int>(centerX + static_cast<float>(dx_offset));
    int ny = static_cast<int>(centerY + static_cast<float>(dy_offset));
    int nz = static_cast<int>(centerZ + static_cast<float>(dz_offset));

    if (nx < 0 || nx >= width || ny < 0 || ny >= height || nz < 0 || nz >= depth) {
        return;
    }

    float weight = computeGaussianWeightForOffset(dx_offset, dy_offset, dz_offset, sigma);

    if (weight > 1e-6f) { // Add only if there's a non-negligible contribution
        int target_idx = IX_device(nx, ny, nz, width, height);
        atomicAdd(&d_Vx[target_idx], amountVx * weight);
        atomicAdd(&d_Vy[target_idx], amountVy * weight);
        atomicAdd(&d_Vz[target_idx], amountVz * weight);
    }
}

// Add velocity with a Gaussian distribution
void FluidSimulation::addVelocityGaussian(float centerX, float centerY, float centerZ, float vx, float vy, float vz, float sigma) {
    if (sigma < 0.0f) {
        sigma = 0.0f;
    }

    int radius = static_cast<int>(sigma * 2.0f);
    if (radius < 0) radius = 0;

    int span = 2 * radius + 1;

    dim3 threadsPerBlock(8, 8, 4); // Example block size, can be tuned
    dim3 numBlocks(
        (span + threadsPerBlock.x - 1) / threadsPerBlock.x,
        (span + threadsPerBlock.y - 1) / threadsPerBlock.y,
        (span + threadsPerBlock.z - 1) / threadsPerBlock.z
    );

    if (numBlocks.x == 0) numBlocks.x = 1;
    if (numBlocks.y == 0) numBlocks.y = 1;
    if (numBlocks.z == 0) numBlocks.z = 1;

    addVelocityGaussianKernel<<<numBlocks, threadsPerBlock>>>(
        d_Vx, d_Vy, d_Vz,
        centerX, centerY, centerZ,
        vx, vy, vz, sigma, radius,
        params.width, params.height, params.depth
    );
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());
}

